#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
//#include <map>
#include <iostream>
#include "multiGPU_topology_optimize.cuh"


//#include <hip/hip_runtime_api.h>

using namespace std;

const char *sSampleName = "MultiGPU Latency Matrix analysis and Topology Optimization (Units in microseconds.)";

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

__global__ void delay(int * null) {
  float j=threadIdx.x;
  for(int i=1;i<10000;i++)
      j=(j+1)/j;

  if(threadIdx.x == j) null[0] = j;
}



//Sorting 1D Optimized topology
std::vector<int> getOptimizedTopology1(vector<vector <float> > devices)
{
	vector<int> sorted;
	sorted.resize(devices.size());
	
	//Set availability flags
	vector<int> used;
	used.resize(devices.size());
	
	
	for(int k=0;k<static_cast<int>(devices.size());k++)
	{
		used[k]=0;
	}
	
	//First device is device 0 as the starting point
	sorted[0]=0;
	used[0]=1;

	int currentDevice=0;
		
	for(int i=0;i<static_cast<int>(devices.size()-1);i++)
	{
		//Loop over Each device latency data and find the next device with the lowest  average bidirectional latency
		float lowestLatency=999999.99;
		int nextDevice=-1;
		//j is a placeHolder for the next device
		for(int j=0;j<static_cast<int>(devices[i].size());j++)
		{
			//Dont check for the latency on itself
			if(currentDevice != j && used[j]==0)
			{
				float latencyD2D_outgoing=devices[currentDevice][j];
				float latencyD2D_incoming=devices[j][currentDevice];
				
				//Average current Latency less than the previous latency
				if(((latencyD2D_outgoing+latencyD2D_incoming)/2)<lowestLatency)
				{
					lowestLatency=(latencyD2D_outgoing+latencyD2D_incoming)/2;
					nextDevice=j;
				}
				
			}
		}
		//add to the Device list
		sorted[i+1]=nextDevice;
		//Prevent device reassign in future slots
		used[nextDevice]=1;
		currentDevice=nextDevice;
					
		
	}
		
	return sorted;
}

//Sorting 2D Optimized topology
std::vector<int> getOptimizedTopology2(vector<vector<float> > devices, int numberOfDevicesAlong_X, int numberOfDevicesAlong_Y)
{
	vector<int> sorted;
	sorted.resize(devices.size());
	
	
	//Set availability flags
	vector<int> used;
	used.resize(devices.size());
	for(int k=0;k<static_cast<int>(devices.size());k++)
	{
		used[k]=0;
	}
	
	
	//First device is device 0 as the starting point
	sorted[0]=0;
	used[0]=1;
	//Considering Left and Bottom devices  Averages Bidirectional Latency calculations: In 3D bottom, front and left would be considered
				
	int currentDeviceLeft=0;
	int currentDeviceBottom=0;
	

	
	for(int i=0;i<static_cast<int>(devices.size()-1);i++)
	{
		//Loop over Each device latency data and find the next device with the lowest  average bidirectional latency
		
		float lowestLatencyLeft=999999.99;
		float lowestLatencyBottom=999999.99;
		float combinedLowestLatency=999999.99*2;
		
		int nextDevice=-1;
		//j is a placeHolder for the next device
		for(int j=0;j<static_cast<int>(devices[i].size());j++)
		{
			//Dont check for the latency on itself
			if(currentDeviceLeft != j && currentDeviceBottom != j && used[j]==0)
			{
				//Left
				//Ignoring the first column : Where there is no left device
				if((i%numberOfDevicesAlong_X)!=0)
				{
					float l_latencyD2D_outgoing=devices[currentDeviceLeft][j];
					float l_latencyD2D_incoming=devices[j][currentDeviceLeft];
					
					//Average current Latency less than the previous latency
					if(((l_latencyD2D_outgoing+l_latencyD2D_incoming)/2)<lowestLatencyLeft)
					{
						lowestLatencyLeft=(l_latencyD2D_outgoing+l_latencyD2D_incoming)/2;
						
					}
				
				}
				
				//Bottom
				//Ignoring the last row: Where there is no bottom Device
				if(i>=numberOfDevicesAlong_X)
				{
					
					float b_latencyD2D_outgoing=devices[currentDeviceBottom][j];
					float b_latencyD2D_incoming=devices[j][currentDeviceBottom];
					
					//Average current Latency less than the previous latency
					if(((b_latencyD2D_outgoing+b_latencyD2D_incoming)/2)<lowestLatencyBottom)
					{
						lowestLatencyBottom=(b_latencyD2D_outgoing+b_latencyD2D_incoming)/2;
						
					}
					
					
				}
				
				if(((lowestLatencyBottom+lowestLatencyLeft)/2)<combinedLowestLatency)
				{
					combinedLowestLatency=(lowestLatencyBottom+lowestLatencyLeft)/2;
					nextDevice=j;
				}
				
				
				
				
			}
		}
		//add to the Device list
		sorted[i+1]=nextDevice;
		used[nextDevice]=1;
		
		
		//Set Device Left
		if((i%numberOfDevicesAlong_X)!=0)
		{
			currentDeviceLeft=nextDevice;
		}
		
		//Set Device Bottom
		if(i>=numberOfDevicesAlong_X)
		{
			currentDeviceBottom=sorted[(i-numberOfDevicesAlong_X)+1];
		}			
			
	}
	return sorted;
}




map<int,int> outputLatencyMatrix(int numGPUs, bool p2p, int numberOfDevicesAlong_X, int numberOfDevicesAlong_Y, int domainDecom_Dim)
{
	cout<<endl<<sSampleName<<endl;
    int repeat=10000;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],1);
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> latencyMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if(p2p) {
                hipDeviceCanAccessPeer(&access,i,j);
                if (access)
                {
                    hipDeviceEnablePeerAccess(j,0);
                    cudaCheckError();
                }
            }
            hipDeviceSynchronize();
            cudaCheckError();
            delay<<<1,1>>>(NULL);
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,1);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);

            latencyMatrix[i*numGPUs+j]=time_ms*1e3/repeat;
            if(p2p && access)
            {
                hipDeviceDisablePeerAccess(j);
            }
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

	
	//Each Device stores the latency vector to other devices
	std::vector< vector<float> >device(numGPUs, vector<float> (numGPUs));

    for (int i=0; i<numGPUs; i++)
    {
		//Select GPUs here.
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
			//Add latency to other GPUs here
			device[i][j]=latencyMatrix[i*numGPUs+j];
			
            printf("%6.02f ", device[i][j]);
        }

        printf("\n");
    }
	
	//Test matrix 8x8 for 8 GPUs
	
	//P2P=Enabled Latency Matrix (us)
	//D\D     0      1      2      3      4      5      6      7 
     //0   6.18  21.22  21.36  25.07  37.26  41.03  26.83  41.13 
     //1  27.48   6.09  21.28  24.82  37.67  40.89  26.81  40.91 
     //2  27.53  21.44   6.13  24.95  37.73  40.88  26.53  40.92 
     //3  27.51  21.38  21.16   5.77  38.09  41.10  27.16  41.10 
     //4  40.41  33.29  33.10  36.86   6.45  24.77   9.97  28.11 
     //5  40.42  33.15  32.84  36.62  24.83   6.07   9.93  28.17 
     //6  39.25  33.27  32.99  37.13  24.93  24.93   6.07  28.08 
     //7  40.48  33.66  33.53  36.75  24.85  24.90  10.05   6.37
	 
	 
	 //2D - 4 x 2 order should be
	 
	 
	 
	 
	/* std::vector< vector<float> >deviceTest(8, vector<float> (8));
	 
	 
	 
	 deviceTest[0][0]=6.18;
	 deviceTest[0][1]=21.22;
	 deviceTest[0][2]=21.36;
	 deviceTest[0][3]=25.07;
	 deviceTest[0][4]=37.26;
	 deviceTest[0][5]=41.03;
	 deviceTest[0][6]=26.83;
	 deviceTest[0][7]=41.13;
	 
	 deviceTest[1][0]=27.48;
	 deviceTest[1][1]=6.09;
	 deviceTest[1][2]=21.28;
	 deviceTest[1][3]=24.82;
	 deviceTest[1][4]=37.67;
	 deviceTest[1][5]=40.89;
	 deviceTest[1][6]=26.81;
	 deviceTest[1][7]=40.91;
	 
	 deviceTest[2][0]=27.53;
	 deviceTest[2][1]= 21.44;
	 deviceTest[2][2]=6.13;
	 deviceTest[2][3]=24.95;
	 deviceTest[2][4]=37.73;
	 deviceTest[2][5]=40.88;
	 deviceTest[2][6]=26.53;
	 deviceTest[2][7]=40.92;
	 
	 deviceTest[3][0]=27.51;
	 deviceTest[3][1]=21.38;
	 deviceTest[3][2]=21.16;
	 deviceTest[3][3]=5.77;
	 deviceTest[3][4]=38.09;
	 deviceTest[3][5]=41.10;
	 deviceTest[3][6]=27.16;
	 deviceTest[3][7]=41.10;
	 
	 
	 //4  40.41  33.29  33.10  36.86   6.45  24.77   9.97  28.11 
     //5  40.42  33.15  32.84  36.62  24.83   6.07   9.93  28.17 
     //6  39.25  33.27  32.99  37.13  24.93  24.93   6.07  28.08 
     //7  40.48  33.66  33.53  36.75  24.85  24.90  10.05   6.37
	 
	 deviceTest[4][0]=40.41;
	 deviceTest[4][1]=33.29;
	 deviceTest[4][2]=33.10;
	 deviceTest[4][3]=36.86;
	 deviceTest[4][4]=6.45;
	 deviceTest[4][5]=24.77;
	 deviceTest[4][6]=9.97;
	 deviceTest[4][7]=28.11;
	 
	 deviceTest[5][0]=40.42;
	 deviceTest[5][1]=33.15;
	 deviceTest[5][2]=32.84;
	 deviceTest[5][3]=36.62;
	 deviceTest[5][4]=24.83;
	 deviceTest[5][5]=6.07;
	 deviceTest[5][6]=9.93;
	 deviceTest[5][7]=28.17;
	 
	 deviceTest[6][0]=39.25;
	 deviceTest[6][1]=33.27;
	 deviceTest[6][2]=32.99;
	 deviceTest[6][3]=37.13;
	 deviceTest[6][4]=24.93;
	 deviceTest[6][5]=24.93;
	 deviceTest[6][6]=6.07;
	 deviceTest[6][7]=28.08;
	 
	 
	 deviceTest[7][0]= 40.48;
	 deviceTest[7][1]=33.66;
	 deviceTest[7][2]=33.53;
	 deviceTest[7][3]=36.75;
	 deviceTest[7][4]=24.85;
	 deviceTest[7][5]=24.90;
	 deviceTest[7][6]=10.05;
	 deviceTest[7][7]=6.37;*/
	
	 
	
	
	//Sort The Devices according to best performance
	map<int,int> optimized;
	//Initialize
	for(int init=0;init<numGPUs;init++)
	{
		optimized[init]=init;
	}
	
	
	
	if(domainDecom_Dim==1)
	{
		vector<int> getSortOrder_1D=getOptimizedTopology1(device);
		cout<<"1D Sort Order"<<endl;
	
		for(int j=0;j<static_cast<int>(getSortOrder_1D.size());j++)
		{
			//Dont check for the latency on itself
			cout<<"Device "<<j+1<<"   "<<getSortOrder_1D[j]<<endl;
			optimized[j]=getSortOrder_1D[j];
		}
	}
	else//For 2D domain and higher
	{
		vector<int> getSortOrder_2D=getOptimizedTopology2(device,numberOfDevicesAlong_X,numberOfDevicesAlong_Y);
		cout<<"2D Sort Order"<<endl;
		for(int j=0;j<static_cast<int>(getSortOrder_2D.size());j++)
		{
			//Dont check for the latency on itself
			cout<<"Device "<<j+1<<"   "<<getSortOrder_2D[j]<<endl;
			optimized[j]=getSortOrder_2D[j];
		}
	}
	
	
	
	
		

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
	
	
	
	return optimized;
}

/*int main(int argc, char **argv)
{

    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    printf("[%s]\n", sSampleName);

    //output devices
    for (int i=0; i<numGPUs; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n",i,prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
    }

    checkP2Paccess(numGPUs);

    //Check peer-to-peer connectivity
    printf("P2P Connectivity Matrix\n");
    printf("     D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d", j);
    }
    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d\t", i);
        for (int j=0; j<numGPUs; j++)
        {
            if (i!=j)
            {
               int access;
               hipDeviceCanAccessPeer(&access,i,j);
               printf("%6d", (access) ? 1 : 0);
            }
            else
            {
                printf("%6d", 1);
            }
        }
        printf("\n");
    }

 

    printf("P2P=Enabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs, true);

   
    exit(EXIT_SUCCESS);
}*/
